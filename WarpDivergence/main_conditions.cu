#include <hip/hip_runtime.h>


#include <cstdlib>
#include <iostream>

__global__ void DivergencyKernel(float* a, int N) {
  int x = blockDim.x * blockIdx.x + threadIdx.x;

  if (!(threadIdx.x % 2))
    a[x] = a[x] * (threadIdx.x + 1);
  else
    a[x] = a[x] * (threadIdx.x % 5);
}

__global__ void NoDivergencyKernel(float* a, int N) {
  int x = blockDim.x * blockIdx.x + threadIdx.x;
  a[x] = threadIdx.x;
}

int main(int argc, char** argv) {
  if (argc == 2) {
    int N = atoi(argv[1]);
    size_t size = N * sizeof(float);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float* h_A = (float*)malloc(size);
    if (h_A == NULL) {
      std::cerr << "Failed malloc for h_A!\n";
      return 1;
    }

    for (int i = 0; i < N; i++) {
      h_A[i] = i + 1;
    }

    float* d_A = NULL;
    hipMalloc((void**)&d_A, size);
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    const int BLOCK_SIZE = 1024;
    const int GRID_SIZE = (N - 1) / BLOCK_SIZE + 1;
    hipEventRecord(start);
    DivergencyKernel<<<BLOCK_SIZE, GRID_SIZE>>>(d_A, N);
    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float msecs = 0;
    hipEventElapsedTime(&msecs, start, stop);
    std::cout << "(Divergency) Kernel Time: " << msecs << " ms.\n";

    hipEventRecord(start);
    NoDivergencyKernel<<<BLOCK_SIZE, GRID_SIZE>>>(d_A, N);
    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&msecs, start, stop);
    std::cout << "(Non-Divergency) Kernel Time: " << msecs << " ms.\n";

    hipFree(d_A);
    free(h_A);
  }

  return 0;
}