#include <hip/hip_runtime.h>


#include <chrono>
#include <cstdlib>
#include <iostream>

__global__ void addVectorsKernel(const double* a,
                                 const double* b,
                                 double* c,
                                 int n) {
  int x = blockDim.x * blockIdx.x + threadIdx.x;
  while (x < n) {
    c[x] = a[x] + b[x];
    printf("x = %d | threadIdx.x = %d | blockIdx.x = %d\n", x, threadIdx.x,
           blockIdx.x);
  }
}

int main(int argc, char** argv) {
  if (argc == 2) {
    int n = atoi(argv[1]);
    size_t size = n * sizeof(double);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    double* h_A = (double*)malloc(size);
    double* h_B = (double*)malloc(size);
    double* h_C = (double*)malloc(size);

    for (int i = 0; i < n; i++) {
      h_A[i] = rand() % 10;
      h_B[i] = rand() % 10;
    }

    double* d_A = NULL;
    double* d_B = NULL;
    double* d_C = NULL;

    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    double Blocks = 1024;                 // threads per block
    double Grids = (n - 1) / Blocks + 1;  // blocks per grid

    hipEventRecord(start);
    addVectorsKernel<<<Grids, Blocks>>>(d_A, d_B, d_C, n);
    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);
    float msecs = 0;
    hipEventElapsedTime(&msecs, start, stop);
    std::cout << "GPU Elapsed Time: " << msecs << " ms.\n";

    for (int i = 0; i < n; i++) {
      if (h_C[i] != h_A[i] + h_B[i]) {
        std::cerr << "TEST FAILED...\n";
        return 1;
      }
    }

    std::cout << "TEST PASSED!\n";

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);
  }

  return 0;
}