//
//  main.cpp
//
//
//  Created by Elijah Afanasiev on 25.09.2018.
//
//

// System includes
#include <assert.h>
#include <stdio.h>
#include <chrono>
#include <cstdlib>
#include <iostream>

// CUDA runtime
#include <hip/hip_runtime.h>



#ifndef MAX
#define MAX(a, b) (a > b ? a : b)
#endif

__global__ void vectorAddGPU(float* a, float* b, float* c, int N, int offset) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < N) {
    c[offset + idx] = a[offset + idx] + b[offset + idx];
  }
}

void sample_vec_add(int size = 1048576) {
  int n = size;
  int nBytes = n * sizeof(int);

  float *a, *b, *c;

  a = (float*)malloc(nBytes);
  b = (float*)malloc(nBytes);
  c = (float*)malloc(nBytes);

  float *d_A, *d_B, *d_C;

  dim3 block(256);
  dim3 grid((unsigned int)ceil(n / (float)block.x));

  for (int i = 0; i < n; i++) {
    a[i] = rand() / (float)RAND_MAX;
    b[i] = rand() / (float)RAND_MAX;
    c[i] = 0;
  }

  hipMalloc((void**)&d_A, n * sizeof(float));
  hipMalloc((void**)&d_B, n * sizeof(float));
  hipMalloc((void**)&d_C, n * sizeof(float));

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);

  hipMemcpy(d_A, a, n * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_B, b, n * sizeof(float), hipMemcpyHostToDevice);

  vectorAddGPU<<<grid, block>>>(d_A, d_B, d_C, n, 0);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("Streams Used: 0\nGPU Elapsed time: %f ms\n", milliseconds);

  hipDeviceSynchronize();

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}

void streams_vec_add(int size = 1048576) {
  int n = size;
  int nBytes = n * sizeof(float);
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  float *a, *b, *c;

  hipHostAlloc((void**)&a, nBytes, hipHostMallocDefault);
  hipHostAlloc((void**)&b, nBytes, hipHostMallocDefault);
  hipHostAlloc((void**)&c, nBytes, hipHostMallocDefault);

  float *d_A, *d_B, *d_C;

  for (int i = 0; i < n; i++) {
    a[i] = rand() / (float)RAND_MAX;
    b[i] = rand() / (float)RAND_MAX;
    c[i] = 0;
  }

  hipMalloc((void**)&d_A, nBytes);
  hipMalloc((void**)&d_B, nBytes);
  hipMalloc((void**)&d_C, nBytes);

  hipEventRecord(start);

  const int stream_count = 4;
  const int stream_size = n / stream_count;
  hipStream_t Stream[stream_count];
  for (int i = 0; i < stream_count; i++)
    hipStreamCreate(&Stream[i]);

  dim3 block(1024);
  dim3 grid((stream_size - 1) / 1024 + 1);

  for (int i = 0; i < stream_count; i++) {
    int offset = i * stream_size;
    hipMemcpyAsync(&d_A[offset], &a[offset], stream_size * sizeof(float),
                    hipMemcpyHostToDevice, Stream[i]);
    hipMemcpyAsync(&d_B[offset], &b[offset], stream_size * sizeof(float),
                    hipMemcpyHostToDevice, Stream[i]);
    hipMemcpyAsync(&d_C[offset], &c[offset], stream_size * sizeof(float),
                    hipMemcpyHostToDevice, Stream[i]);

    vectorAddGPU<<<grid, block>>>(d_A, d_B, d_C, stream_size, offset);

    hipMemcpyAsync(&a[offset], &d_A[offset], stream_size * sizeof(float),
                    hipMemcpyDeviceToHost, Stream[i]);
    hipMemcpyAsync(&b[offset], &d_B[offset], stream_size * sizeof(float),
                    hipMemcpyDeviceToHost, Stream[i]);
    hipMemcpyAsync(&c[offset], &d_C[offset], stream_size * sizeof(float),
                    hipMemcpyDeviceToHost, Stream[i]);
  }

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float msecs = 0;
  hipEventElapsedTime(&msecs, start, stop);
  std::cout << "Streams Used: " << stream_count
            << "\nGPU Elapsed Time : " << msecs << " ms.\n";

  hipDeviceSynchronize();

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  hipHostFree(a);
  hipHostFree(b);
  hipHostFree(c);
}

int main(int argc, char** argv) {
  sample_vec_add(atoi(argv[1]));

  std::cout << "---------------------\n---------------------\n";
  streams_vec_add(atoi(argv[1]));

  return 0;
}
