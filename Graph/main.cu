
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cfloat>
#include <chrono>
#include <iostream>

using namespace std;

///////////////////////////////////////////////////////////////////////////////////////////////////////////

hipError_t SAFE_CALL(hipError_t result) {
  if (result != hipSuccess) {
    printf("CUDA error: %s at call #CallInstruction\n",
           hipGetErrorString(result));
    throw "error in CUDA API function, aborting...";
  }
  return result;
}

hipError_t SAFE_KERNEL_CALL(hipError_t result) {
  if (result != hipSuccess) {
    printf("CUDA error in kernel launch: %s at kernel #KernelCallInstruction\n",
           hipGetErrorString(result));
    throw "error in CUDA kernel launch, aborting...";
  }
  result = hipDeviceSynchronize();
  if (result != hipSuccess) {
    printf(
        "CUDA error in kernel execution: %s at kernel "
        "\"#KernelCallInstruction\"\n",
        hipGetErrorString(result));
    throw "error in CUDA kernel execution, aborting...";
  }
  return result;
};

///////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void gather(int* ptrs,
                       int* connections,
                       int* out_ids,
                       int vertices_count,
                       int* data,
                       int* result) {
  const long long src_id = (blockIdx.x * blockDim.x + threadIdx.x) / 32;

  if (src_id < vertices_count) {
    const int first_edge_ptr = ptrs[src_id];
    const int cn_count = connections[src_id];
    int tid = threadIdx.x;
    int warp_size = 32;
    for (register int cur_edge = tid % 32; cur_edge < cn_count;
         cur_edge += warp_size) {
      int dst_id = out_ids[first_edge_ptr + cur_edge];
      int val = data[dst_id];
      result[first_edge_ptr + cur_edge] = val;
    }
  }
}
/*
void __global__ gather(int* ptrs,
                       int* connections,
                       int* outgoing_ids,
                       int vertices_count,
                       int* data,
                       int* result) {
  const long long src_id = blockIdx.x * blockDim.x + threadIdx.x;

  if (src_id < vertices_count) {
    const int first_edge_ptr = ptrs[src_id];
    const int connections_count = connections[src_id];
    // connections_count = ptrs[src_id + 1] - ptrs[src_id];

    for (register int cur_edge = 0; cur_edge < connections_count; cur_edge++) {
      // first_edge_ptr + cur_edge - индекс текущего ребра в массивах
      int dst_id = outgoing_ids[first_edge_ptr + cur_edge];
      int val = data[dst_id];
      result[first_edge_ptr + cur_edge] = val;
    }
  }
}
*/

int main() {
  int vertices_count = 1024 * 1024;

  int* ptrs = new int[vertices_count];
  int* data = new int[vertices_count];
  int* connections = new int[vertices_count];

  int pos = 0;
  for (int i = 0; i < vertices_count; i++) {
    ptrs[i] = pos;
    connections[i] = 16 + rand() % 32;
    pos += connections[i];

    data[i] = rand();
  }

  int edges_count = pos;
  int* outgoing_ids = new int[edges_count];
  int* result = new int[edges_count];
  for (int i = 0; i < edges_count; i++) {
    outgoing_ids[i] = rand() % vertices_count;
  }

  int* dev_ptrs;
  int* dev_connections;
  int* dev_outgoing_ids;
  int* dev_data;
  int* dev_result;
  hipMalloc((void**)&dev_ptrs, vertices_count * sizeof(int));
  hipMalloc((void**)&dev_connections, vertices_count * sizeof(int));
  hipMalloc((void**)&dev_data, vertices_count * sizeof(int));
  hipMalloc((void**)&dev_outgoing_ids, edges_count * sizeof(int));
  hipMalloc((void**)&dev_result, edges_count * sizeof(int));

  SAFE_CALL(hipMemcpy(dev_ptrs, ptrs, vertices_count * sizeof(int),
                       hipMemcpyHostToDevice));
  SAFE_CALL(hipMemcpy(dev_connections, connections,
                       vertices_count * sizeof(int), hipMemcpyHostToDevice));
  SAFE_CALL(hipMemcpy(dev_data, data, vertices_count * sizeof(int),
                       hipMemcpyHostToDevice));
  SAFE_CALL(hipMemcpy(dev_outgoing_ids, outgoing_ids,
                       edges_count * sizeof(int), hipMemcpyHostToDevice));

  dim3 compute_threads(1024);
  dim3 compute_blocks((vertices_count - 1) / compute_threads.x + 1);

  for (int i = 0; i < 5; i++) {
    auto start = std::chrono::steady_clock::now();
    gather<<<compute_blocks, compute_threads>>>(
        dev_ptrs, dev_connections, dev_outgoing_ids, vertices_count, dev_data,
        dev_result);
    auto end = std::chrono::steady_clock::now();  // TODO почему работает данный
                                                  // замер веремени?
    std::chrono::duration<double> elapsed_seconds = end - start;
    cout << "time: " << (elapsed_seconds.count()) * 1000.0 << " ms" << endl;
    cout << "bandwidth: "
         << 3.0 * sizeof(int) * edges_count / ((elapsed_seconds.count()) * 1e9)
         << " GB/s" << endl
         << endl;
  }

  int* copy_device_result = new int[edges_count];

  SAFE_CALL(hipMemcpy(copy_device_result, dev_result,
                       edges_count * sizeof(int), hipMemcpyDeviceToHost));
  for (int src_id = 0; src_id < vertices_count; src_id++) {
    const int first_edge_ptr = ptrs[src_id];
    const int connections_count = connections[src_id];

    for (register int cur_edge = 0; cur_edge < connections_count; cur_edge++) {
      int dst_id = outgoing_ids[first_edge_ptr + cur_edge];
      int val = data[dst_id];
      result[first_edge_ptr + cur_edge] = val;
    }
  }

  int errors_count = 0;
  for (int i = 0; i < edges_count; i++) {
    if (result[i] != copy_device_result[i])
      errors_count++;
  }
  cout << errors_count << endl;
  hipFree(dev_data);
  hipFree(dev_ptrs);
  hipFree(dev_connections);
  hipFree(dev_result);
  hipFree(dev_outgoing_ids);

  delete[] result;
  delete[] data;
  delete[] ptrs;
  delete[] outgoing_ids;
  delete[] connections;

  return 0;
}