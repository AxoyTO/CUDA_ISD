#include "hip/hip_runtime.h"
//
//  main.cpp
//
//
//  Created by Elijah Afanasiev on 25.09.2018.
//
//

// System includes
#include <assert.h>
#include <stdio.h>
#include <chrono>
#include <cstdlib>
#include <iostream>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#ifndef MAX
#define MAX(a, b) (a > b ? a : b)
#endif

__global__ void vectorAddGPU(float* a, float* b, float* c, int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < N) {
    c[idx] = a[idx] + b[idx];
  }
}

void unified_sample(int size = 1048576) {
  int n = size;
  int nBytes = n * sizeof(float);

  float *a, *b, *c;
  // float *d_a, *d_b, *d_c;

  hipEvent_t unifiedStart, unifiedStop;
  hipEventCreate(&unifiedStart);
  hipEventCreate(&unifiedStop);

  dim3 block(256);
  dim3 grid((unsigned int)ceil(n / (float)block.x));
  printf("Allocating managed(unified) memory on both host and device..\n");

  hipMallocManaged(&a, nBytes);
  hipMallocManaged(&b, nBytes);
  hipMallocManaged(&c, nBytes);

  for (int i = 0; i < n; i++) {
    a[i] = rand() / (float)RAND_MAX;
    b[i] = rand() / (float)RAND_MAX;
  }

  hipEventRecord(unifiedStart);
  printf("Doing GPU Vector add\n");
  vectorAddGPU<<<grid, block>>>(a, b, c, n);
  hipEventRecord(unifiedStop);
  hipDeviceSynchronize();

  float elapsedUnified;
  hipEventElapsedTime(&elapsedUnified, unifiedStart, unifiedStop);
  std::cout << "Unified-Memory copying Elapsed Time: " << elapsedUnified
            << " ms.\n";

  hipDeviceSynchronize();
}

void pinned_sample(int size = 1048576) {
  int n = size;
  int nBytes = n * sizeof(float);

  float *h_a, *h_b, *h_c;
  float *d_a, *d_b, *d_c;

  hipEvent_t pinnedStart, pinnedStop;
  hipEventCreate(&pinnedStart);
  hipEventCreate(&pinnedStop);

  dim3 block(256);
  dim3 grid((unsigned int)ceil(n / (float)block.x));

  printf("Allocating device pinned memory on host\n");
  hipHostMalloc(&h_a, nBytes);
  hipHostMalloc(&h_b, nBytes);
  hipHostMalloc(&h_c, nBytes);
  hipMalloc(&d_a, nBytes);
  hipMalloc(&d_b, nBytes);
  hipMalloc(&d_c, nBytes);

  for (int i = 0; i < n; i++) {
    h_a[i] = rand() / (float)RAND_MAX;
    h_b[i] = rand() / (float)RAND_MAX;
    h_c[i] = 0;
  }

  printf("Copying to device..\n");
  printf("Doing GPU Vector Add\n");
  hipEventRecord(pinnedStart);
  hipMemcpy(d_a, h_a, nBytes, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, nBytes, hipMemcpyHostToDevice);
  vectorAddGPU<<<grid, block>>>(d_a, d_b, d_c, n);

  hipEventRecord(pinnedStop);
  hipDeviceSynchronize();

  float elapsedPinned;
  hipEventElapsedTime(&elapsedPinned, pinnedStart, pinnedStop);
  std::cout << "Pinned-Memory copying Elapsed Time: " << elapsedPinned
            << " ms.\n";

  hipDeviceSynchronize();
}

void usual_sample(int size = 1048576) {
  int n = size;

  int nBytes = n * sizeof(float);

  float *a, *b;  // host data
  float* c;      // results

  a = (float*)malloc(nBytes);
  b = (float*)malloc(nBytes);
  c = (float*)malloc(nBytes);

  float *a_d, *b_d, *c_d;

  dim3 block(256);
  dim3 grid((unsigned int)ceil(n / (float)block.x));

  for (int i = 0; i < n; i++) {
    a[i] = rand() / (float)RAND_MAX;
    b[i] = rand() / (float)RAND_MAX;
    c[i] = 0;
  }

  printf("Allocating device memory on host..\n");

  hipMalloc((void**)&a_d, n * sizeof(float));
  hipMalloc((void**)&b_d, n * sizeof(float));
  hipMalloc((void**)&c_d, n * sizeof(float));

  printf("Copying to device..\n");

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);

  hipMemcpy(a_d, a, n * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(b_d, b, n * sizeof(float), hipMemcpyHostToDevice);

  printf("Doing GPU Vector add\n");

  vectorAddGPU<<<grid, block>>>(a_d, b_d, c_d, n);

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("time: %f ms\n", milliseconds);

  hipDeviceSynchronize();

  hipFree(a_d);
  hipFree(b_d);
  hipFree(c_d);
  free(a);
  free(b);
  free(c);
}

int main(int argc, char** argv) {
  std::cout << "-------> USUAL SAMPLE <-------\n";
  usual_sample(atoi(argv[1]));
  std::cout << "-------> USUAL SAMPLE <-------\n\n";
  std::cout << "-------> PINNED SAMPLE <-------\n";
  pinned_sample(atoi(argv[1]));
  std::cout << "-------> PINNED SAMPLE <-------\n\n";
  std::cout << "-------> UNIFIED SAMPLE <-------\n";
  unified_sample(atoi(argv[1]));
  std::cout << "-------> UNIFIED SAMPLE <-------\n";
  return 0;
}