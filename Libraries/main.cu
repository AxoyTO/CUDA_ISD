#include "hip/hip_runtime.h"
// High level matrix multiplication on GPU using CUDA with Thrust, CURAND and
// CUBLAS C(m,n) = A(m,k) * B(k,n)
#include <hipblas.h>
#include <hiprand.h>
#include <cstdlib>
#include <ctime>
#include <iostream>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

// Fill the array A(nr_rows_A, nr_cols_A) with random numbers on GPU
void GPU_fill_rand(float* A, int nr_rows_A, int nr_cols_A) {
  // Create a pseudo-random number generator
  hiprandGenerator_t prng;
  hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

  // Set the seed for the random number generator using the system clock
  hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long)clock());

  // Fill the array with random numbers on the device
  hiprandGenerateUniform(prng, A, nr_rows_A * nr_cols_A);
}

// Multiply the arrays A and B on GPU and save the result in C
// C(m,n) = A(m,k) * B(k,n)
void gpu_blas_mmul(const float* A,
                   const float* B,
                   float* C,
                   const int m,
                   const int k,
                   const int n) {
  int lda = m, ldb = k, ldc = m;
  const float alf = 1;
  const float bet = 0;
  const float* alpha = &alf;
  const float* beta = &bet;

  // Create a handle for CUBLAS
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  // Do the actual multiplication
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb,
              beta, C, ldc);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float msecs = 0;
  hipEventElapsedTime(&msecs, start, stop);
  std::cout << "cublasSGEMM Elapsed Time on GPU: " << msecs << " ms.\n";
  float numOps = 2 * 3 * 3;
  std::cout << "Efficiency of the program: " << numOps / (msecs * 1000)
            << " GFLOPS.\n\n";

  // Destroy the handle
  hipblasDestroy(handle);
}

// Print matrix A(nr_rows_A, nr_cols_A) storage in column-major format
void print_matrix(const thrust::device_vector<float>& A,
                  int nr_rows_A,
                  int nr_cols_A) {
  for (int i = 0; i < nr_rows_A; ++i) {
    for (int j = 0; j < nr_cols_A; ++j) {
      std::cout << A[j * nr_rows_A + i] << " ";
    }
    std::cout << std::endl;
  }
  std::cout << std::endl;
}

int main() {
  // Allocate 3 arrays on CPU
  int nr_rows_A, nr_cols_A, nr_rows_B, nr_cols_B, nr_rows_C, nr_cols_C;

  // for simplicity we are going to use square arrays
  nr_rows_A = nr_cols_A = nr_rows_B = nr_cols_B = nr_rows_C = nr_cols_C = 3;

  thrust::device_vector<float> d_A(nr_rows_A * nr_cols_A),
      d_B(nr_rows_B * nr_cols_B), d_C(nr_rows_C * nr_cols_C);

  // Fill the arrays A and B on GPU with random numbers
  GPU_fill_rand(thrust::raw_pointer_cast(&d_A[0]), nr_rows_A, nr_cols_A);
  GPU_fill_rand(thrust::raw_pointer_cast(&d_B[0]), nr_rows_B, nr_cols_B);

  // Optionally we can print the data
  std::cout << "A =" << std::endl;
  print_matrix(d_A, nr_rows_A, nr_cols_A);
  std::cout << "B =" << std::endl;
  print_matrix(d_B, nr_rows_B, nr_cols_B);

  // Multiply A and B on GPU
  gpu_blas_mmul(
      thrust::raw_pointer_cast(&d_A[0]), thrust::raw_pointer_cast(&d_B[0]),
      thrust::raw_pointer_cast(&d_C[0]), nr_rows_A, nr_cols_A, nr_cols_B);

  // Print the result
  std::cout << "C =" << std::endl;
  print_matrix(d_C, nr_rows_C, nr_cols_C);

  return 0;
}