// UCSC CMPE220 Advanced Parallel Processing
// Prof. Heiner Leitz
// Author: Marcelo Siero.
// Modified from code by:: Andreas Goetz (agoetz@sdsc.edu)
// CUDA program to perform 1D stencil operation in parallel on the GPU
//
// /* FIXME */ COMMENTS ThAT REQUIRE ATTENTION


#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <stdio.h>
#include <chrono>
#include <iostream>

// define vector length, stencil radius,
#define INPUTSIZE 64l
#define N (1024 * 1024 * INPUTSIZE)
#define RADIUS 3
#define GRIDSIZE 512
#define BLOCKSIZE 1024

int gridSize = GRIDSIZE;
int blockSize = BLOCKSIZE;

float milliseconds = 0;

void cudaErrorCheck() {
  hipError_t error = hipGetLastError();
  const char* errorName = hipGetErrorName(error);
  std::string s0(errorName);
  std::cout << ">>>> " << s0 << std::endl;

  const char* errorDescription = hipGetErrorString(error);
  std::string s1(errorDescription);
  std::cout << ">>>> " << s1 << std::endl;
}

void start_timer(hipEvent_t* start) {
  // FIXME: ADD TIMING CODE, HERE, USE GLOBAL VARIABLES AS NEEDED.
  hipEventCreate(start);
  hipEventRecord(*start);
}

float stop_timer(hipEvent_t* start, hipEvent_t* stop) {
  // FIXME: ADD TIMING CODE, HERE, USE GLOBAL VARIABLES AS NEEDED.
  hipEventCreate(stop);
  hipEventRecord(*stop);

  hipEventSynchronize(*stop);
  hipEventElapsedTime(&milliseconds, *start, *stop);
  return (milliseconds);
}

hipDeviceProp_t prop;
int device;
void getDeviceProperties() {
  /*
  FIXME: Implement this function so as to acquire and print the following
  device properties:
     Major and minor CUDA capability, total device global memory,
     size of shared memory per block, number of registers per block,
     warp size, max number of threads per block, number of multi-prccessors
     (SMs) per device, Maximum number of threads per block dimension (x,y,z),
     Maximumum number of blocks per grid dimension (x,y,z).

  These properties can be useful to dynamically optimize programs.  For
  instance the number of SMs can be useful as a heuristic to determine
  how many is a good number of blocks to use.  The total device global
  memory might be important to know just how much data to operate on at
  once.
  */
}

void newline() {
  std::cout << std::endl;
};

void printThreadSizes() {
  int noOfThreads = gridSize * blockSize;
  printf("Blocks            = %d\n", gridSize);   // no. of blocks to launch.
  printf("Threads per block = %d\n", blockSize);  // no. of threads to launch.
  printf("Total threads     = %d\n", noOfThreads);
  printf("Number of grids   = %d\n", (N + noOfThreads - 1) / noOfThreads);
}
/*
-------------------------------------------------------
CUDA device function that performs 1D stencil operation
-------------------------------------------------------
*/
__global__ void stencil_1D(int* in, int* out, long dim) {
  __shared__ int temp[BLOCKSIZE + 2 * RADIUS];

  long gindex = threadIdx.x + blockDim.x * blockIdx.x;
  int stride = gridDim.x * blockDim.x;
  int tid = threadIdx.x;
  int lindex = threadIdx.x + RADIUS;

  // Go through all data
  // Step all threads in a block to avoid synchronization problem
  while (gindex < dim + blockDim.x) { /* FIXME PART 2 - MODIFY PROGRAM TO USE
                                         SHARED MEMORY. */
    if (gindex < dim) {
      temp[lindex] = in[gindex];
    } else {
      temp[lindex] = 0;
    }

    if (tid < RADIUS) {
      if (gindex < RADIUS) {
        temp[lindex - RADIUS] = 0;
      } else {
        temp[lindex - RADIUS] = in[gindex - RADIUS];
      }
      if (gindex + BLOCKSIZE >= dim) {
        temp[lindex + BLOCKSIZE] = 0;
      } else {
        temp[lindex + BLOCKSIZE] = in[gindex + BLOCKSIZE];
      }
    }

    __syncthreads();

    // Apply the stencil
    int result = 0;
    for (int offset = -RADIUS; offset <= RADIUS; offset++) {
      if (lindex + offset < dim && lindex + offset > -1)
        result += temp[lindex + offset];
    }

    // Store the result
    if (gindex < dim)
      out[gindex] = result;

    // Update global index and quit if we are done
    gindex += stride;

    __syncthreads();
  }
}

#define True 1
#define False 0
void checkResults(
    int* h_in,
    int* h_out,
    int DoCheck =
        True) { /*
                DO NOT CHANGE THIS CODE.
                CPU calculates the stencil from data in *h_in
                if DoCheck is True (default) it compares it with *h_out
                to check the operation of this code.
                If DoCheck is set to False, it can be used to time the CPU.
                */
  int i, j, ij, result, err;
  err = 0;
  for (i = 0; i < N; i++) {  // major index.
    result = 0;
    for (j = -RADIUS; j <= RADIUS; j++) {
      ij = i + j;
      if (ij >= 0 && ij < N)
        result += h_in[ij];
    }
    if (DoCheck) {  // print out some errors for debugging purposes.
      if (h_out[i] != result) {  // count errors.
        err++;
        if (err < 8) {  // help debug
          printf("h_out[%d]=%d should be %d\n", i, h_out[i], result);
        };
      }
    } else {  // for timing purposes.
      h_out[i] = result;
    }
  }

  if (DoCheck) {  // report results.
    if (err != 0) {
      printf("Error, %d elements do not match!\n", err);
    } else {
      printf("Success! All elements match CPU result.\n");
    }
  }
}
/*
------------
main program
------------
*/
int main(void) {
  int *h_in, *h_out;
  int *d_in, *d_out;
  long size = N * sizeof(int);
  int i;
  hipEvent_t start, stop;

  // allocate host memory
  h_in = new int[N];
  h_out = new int[N];

  // getDeviceProperties();

  // initialize vector
  for (i = 0; i < N; i++) {
    //    h_in[i] = i+1;
    h_in[i] = 1;
  }

  // allocate device memory
  hipMalloc((void**)&d_in, size);
  hipMalloc((void**)&d_out, size);
  cudaErrorCheck();

  // copy input data to device
  hipMemcpy(d_in, h_in, size, hipMemcpyHostToDevice);
  cudaErrorCheck();

  // Apply stencil by launching a sufficient number of blocks
  printf("\n---------------------------\n");
  printf("Launching 1D stencil kernel\n");
  printf("---------------------------\n");
  printf("Vector length     = %ld (%ld MB)\n", N, N * 4 / 1024 / 1024);
  printf("Stencil radius    = %d\n", RADIUS);

  //----------------------------------------------------------
  // CODE TO RUN AND TIME THE STENCIL KERNEL.
  //----------------------------------------------------------
  newline();
  printThreadSizes();
  start_timer(&start);
  stencil_1D<<<gridSize, blockSize>>>(d_in, d_out, N);
  std::cout << "Elapsed time: " << stop_timer(&start, &stop) << " ms"
            << std::endl;
  // copy results back to host
  hipMemcpy(h_out, d_out, size, hipMemcpyDeviceToHost);
  cudaErrorCheck();
  checkResults(h_in, h_out);
  //----------------------------------------------------------

  // deallocate device memory
  hipFree(d_in);
  hipFree(d_out);
  cudaErrorCheck();
  //=====================================================
  // Evaluate total time of execution with just the CPU.
  //=====================================================
  newline();
  std::cout << "Running stencil with the CPU.\n";
  start_timer(&start);
  // Use checkResults to time CPU version of the stencil with False flag.
  checkResults(h_in, h_out, False);
  std::cout << "Elapsed time: " << stop_timer(&start, &stop) << " ms"
            << std::endl;
  //=====================================================

  // deallocate host memory
  free(h_in);
  free(h_out);

  return 0;
}